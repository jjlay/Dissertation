/*
 * Simple Monte Carlo
 * Single Threaded CPU Version
 *
 * JJ Lay
 * Middle Tennessee State University
 * October 2014
 *
 * DATE        AUTHOR  COMMENTS
 * ----------  ------  ---------------
 * 2014-10-07  JJL     Initial version
 *
 *
 */


 //
 // CUDA includes
 //

#include "hip/hip_runtime.h"



//
// Function: kernel()
//
// Parameters:
//
// Returns:
//

__global__ void kernel() {

}

