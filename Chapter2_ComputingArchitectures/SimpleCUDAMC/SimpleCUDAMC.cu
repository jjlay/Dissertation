#include "hip/hip_runtime.h"
/*
 * Simple Monte Carlo
 * Single Threaded CPU Version
 *
 * JJ Lay
 * Middle Tennessee State University
 * October 2014
 *
 * DATE        AUTHOR  COMMENTS
 * ----------  ------  ---------------
 * 2014-10-07  JJL     Initial version
 *
 *
 */


//
// CUDA includes
//

#include "hip/hip_runtime.h"
#include ""


//
// Local includes
//

#include "ReturnValues.h"
#include "kernel.cuh"


//
// Standard includes
//

#include <iostream>
#include <random>


//
// Function: main()
//
// Parameters:
//    argc - Number of command line parameters
//    argv[] - Command line parameters
//
// Returns:
//    Completion status
//

int main(int argc, char* argv[]) {

	// Random number

	std::default_random_engine generator;
	std::normal_distribution<double> normal(0, 1);


	// Monte Carlo Parameters

	const int numberSteps = 1000;
	const int numberSimulations = 10000;

	// Black-Scholes parameters

	const double T = 1.0;
	const double r = 0.03;
	const double sigma = 0.03;
	const double S0 = 100.0;

	double dt = T / static_cast<double>(numberSteps);
	double sqrtdt = sqrt(dt);

	double analytical = S0 * exp(r * T);

	double sumS = 0.0;

	for (auto sim = 0; sim < numberSimulations; sim++) {
		auto S = S0;

		for (auto step = 0; step < numberSteps; step++) {
			auto dW = normal(generator) * sqrtdt;
			auto dS = r * S * dt + sigma * S * dW;
			S += dS;
		}

		sumS += S;
	}

	auto ES = sumS / static_cast<double>(numberSimulations);
	double variance = 0.0;

	std::cout << "Simulation results:" << std::endl
		<< "Analytical solution: " << analytical << std::endl
		<< "Simulation: " << ES << std::endl
		<< "Variance: " << variance << std::endl << std::endl
		<< "Error: " << std::scientific << ES - analytical << std::endl;

	return _OKAY_;
}
