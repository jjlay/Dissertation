#include "hip/hip_runtime.h"
//
// Weak vs Strong Error
//
// JJ Lay
// January 2018
//

// Sources:
// https://docs.nvidia.com/cuda/hiprand/device-api-overview.html

//
// CUDA Includes
//

#include "hip/hip_runtime.h"
#include ""
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

//
// Standard Includes
//

#include <iostream>
#include <iomanip>
#include <string>
#include <fstream>

//
// Definitions
//

#define _OKAY_   0
#define _CRASH_  1


//
// Prototypes
//

__global__ void WeakStrong(double *pSum, double *pSum2, double *pSamples,
	double *pError, double *pVariance,
	double pS0, double pK, double pr, double pv, double pT, double pActual,
	unsigned int pSims, unsigned int pSteps, double pEpsilon,
	hiprandState *pStates);

void Crash(std::string pFile, int pLine, hipError_t pCUDAError, std::string pMessage);

//
// Function: main()
//

int main(int argc, char *argv[])
{
	//
	// Monte Carlo Controls
	//

	unsigned int TotalSimsDesired = 1E8;

	double Epsilon = 1E-3;
	unsigned int Steps = 1000;
	int Threads = 100;
	unsigned int Samples = 1 + TotalSimsDesired / Threads;

	//
	// Finance Parameters
	//

	double K = 110.0;   // Strike price
	double S0 = 100.0;  // Initial asset price
	double r = 0.05;    // Interest rate
	double v = 0.5;     // Volatility
	double T = 1.0;     // Time to expiry						
	double Actual = S0 * exp(r * T);  // Per Mathematica's FinancialDerivative function

	//
	// Results
	//

	double Result = 0.0;
	double Error = 0.0;
	double ErrorPercent = 0.0;
	double Variance = 0.0;

	//
	// CUDA Storage
	//

	double *dev_Variance, *host_Variance,
		*dev_Sum, *host_Sum,
		*dev_Sum2, *host_Sum2,
		*dev_Samples, *host_Samples,
		*dev_Error, *host_Error;

	//
	// CUDA Setup
	//

	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		Crash(__FILE__, __LINE__, cudaStatus, "hipSetDevice failed");
	}

	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		Crash(__FILE__, __LINE__, cudaStatus, "hipDeviceReset failed");
	}


	//
	// CUDA Memory Allocation
	//

	cudaStatus = hipMalloc((void**)&dev_Samples, Threads * sizeof(double));
	if (cudaStatus != hipSuccess) {
		Crash(__FILE__, __LINE__, cudaStatus, "dev_Samples hipMalloc failed!");
	}

	cudaStatus = hipMalloc((void**)&dev_Variance, Threads * sizeof(double));
	if (cudaStatus != hipSuccess) {
		Crash(__FILE__, __LINE__, cudaStatus, "dev_Variance hipMalloc failed!");
	}

	cudaStatus = hipMalloc((void**)&dev_Error, Threads * sizeof(double));
	if (cudaStatus != hipSuccess) {
		Crash(__FILE__, __LINE__, cudaStatus, "dev_Error hipMalloc failed!");
	}

	cudaStatus = hipMalloc((void**)&dev_Sum, Threads * sizeof(double));
	if (cudaStatus != hipSuccess) {
		Crash(__FILE__, __LINE__, cudaStatus, "dev_Sum hipMalloc failed!");
	}

	cudaStatus = hipMalloc((void**)&dev_Sum2, Threads * sizeof(double));
	if (cudaStatus != hipSuccess) {
		Crash(__FILE__, __LINE__, cudaStatus, "dev_Sum2 hipMalloc failed!");
	}

	//
	// Local Memory Allocation
	//

	host_Sum = (double *)malloc(Threads * sizeof(double));
	host_Sum2 = (double *)malloc(Threads * sizeof(double));
	host_Error = (double *)malloc(Threads * sizeof(double));
	host_Variance = (double *)malloc(Threads * sizeof(double));
	host_Samples = (double *)malloc(Threads * sizeof(double));

	//
	// Random Number Generator
	//

	hiprandState *dev_States;
	cudaStatus = hipMalloc((void**)&dev_States, Threads * sizeof(hiprandState));
	if (cudaStatus != hipSuccess) {
		Crash(__FILE__, __LINE__, cudaStatus, "dev_States hipMalloc failed!");
	}

	//
	// Log File Initialization
	//

	std::ofstream LogFile("Results.csv", std::ios::app);

	//
	// Run the simulation with smaller and smaller step sizes
	//

	for (auto NumSteps = 1; NumSteps < 1000; NumSteps++) {
		//
		// Kernel Launch
		//

		WeakStrong << <1, Threads >> > (dev_Sum, dev_Sum2, dev_Samples, dev_Error, dev_Variance,
			S0, K, r, v, T, Actual, Samples, NumSteps, Epsilon, dev_States);


		// Check for any errors launching the kernel
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			Crash(__FILE__, __LINE__, cudaStatus, "Kernel launch failed!");
		}

		// hipDeviceSynchronize waits for the kernel to finish, and returns
		// any errors encountered during the launch.
		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			Crash(__FILE__, __LINE__, cudaStatus, "hipDeviceSynchronize failed!");
		}


		//
		// Retrieve Results
		//

		// Copy output vector from GPU buffer to host memory.
		cudaStatus = hipMemcpy(host_Error, dev_Error, Threads * sizeof(double), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			Crash(__FILE__, __LINE__, cudaStatus, "dev_Error Memcpy failed!");
		}

		cudaStatus = hipMemcpy(host_Variance, dev_Variance, Threads * sizeof(double), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			Crash(__FILE__, __LINE__, cudaStatus, "dev_Variance Memcpy failed!");
		}

		cudaStatus = hipMemcpy(host_Sum, dev_Sum, Threads * sizeof(double), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			Crash(__FILE__, __LINE__, cudaStatus, "dev_Sum Memcpy failed!");
		}

		cudaStatus = hipMemcpy(host_Sum2, dev_Sum2, Threads * sizeof(double), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			Crash(__FILE__, __LINE__, cudaStatus, "dev_Sum2 Memcpy failed!");
		}

		cudaStatus = hipMemcpy(host_Samples, dev_Samples, Threads * sizeof(double), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			Crash(__FILE__, __LINE__, cudaStatus, "dev_Samples Memcpy failed!");
		}


		//
		// Export Results
		//


		double FinalSum = 0.0;
		double FinalSamples = 0.0;
		double FinalSum2 = 0.0;
		double FinalSumError = 0.0;
		double FinalStdDev = 0.0;

		for (auto i = 0; i < Threads; i++) {
			FinalSum += host_Sum[i];
			FinalSamples += host_Samples[i];
			FinalSum2 += host_Sum2[i];
			FinalSumError += host_Error[i];
			FinalStdDev += host_Variance[i];
		}

		double FinalMean = FinalSum / FinalSamples;
		double FinalError = FinalMean - Actual;
		double FinalVariance = (FinalSum2 / FinalSamples) - (FinalMean * FinalMean);
		double FinalSumErrorMean = FinalSumError / FinalSamples;
		FinalStdDev = sqrt(FinalStdDev / FinalSamples);

		double dt = T / static_cast<double>(NumSteps);


		std::cout << "Analytical: " << Actual << ", "
			<< "Final Mean: " << FinalMean << ", "
			<< "Final Weak Error: " << std::fixed << std::setprecision(6) << FinalError << ", "
			<< "Final Strong Error: " << FinalSumErrorMean << ", "
			<< "dt: " << dt << ", "
			<< "StdDev: " << FinalStdDev << std::endl;

		LogFile << "Analytical: " << Actual << ", "
			<< "Final Mean: " << FinalMean << ", "
			<< "Final Weak Error: " << std::fixed << std::setprecision(6) << FinalError << ", "
			<< "Final Strong Error: " << FinalSumErrorMean << ", "
			<< "dt: " << dt << ", "
			<< "StdDev: " << FinalStdDev << std::endl;

		LogFile.flush();
	}

	LogFile.close();

	//
	// Memory Cleanup
	//

	free(host_Sum);
	free(host_Sum2);
	free(host_Error);
	free(host_Variance);
	free(host_Samples);

	hipFree(dev_Sum);
	hipFree(dev_Sum2);
	hipFree(dev_Error);
	hipFree(dev_Variance);
	hipFree(dev_Samples);


	//
	// Wrap Up
	//

	return _OKAY_;
}


//
// Kernel: WeakStrong
//

__global__ void WeakStrong(double *pSum, double *pSum2, double *pSamples, 
	double *pError, double *pVariance,
	double pS0, double pK, double pr, double pv, double pT, double pActual,
	unsigned int pSims, unsigned int pSteps, double pEpsilon,
	hiprandState *pStates)
{
	int idx = threadIdx.x;

	hiprand_init(31415, idx, 0, &pStates[idx]);

	double dt = pT / static_cast<double>(pSteps);
	double Sqrtdt = sqrt(dt);

	//
	// Aggregate Variables
	//

	double Sum = 0.0;
	double Sum2 = 0.0;
	double SumError = 0.0;
	double Variance = 0.0;

	//
	// Perform Simulation
	//

	// We use Welford's Algorithm to estimate variance

	// variance(samples) :
	// 	M: = 0
	// 	S : = 0
	// 	for k from 1 to N :	
	// 		x: = samples[k]
	// 		oldM : = M
	// 		M : = M + (x - M) / k
	// 		S : = S + (x - M)*(x - oldM)
	// return S / (N - 1)

	double WM = 0.0, WS = 0.0, WX = 0.0, WMOld = 0.0;  

	for (unsigned int i = 0; i < pSims; i++) {
		double Result = 0.0;
		double S = pS0;

		for (unsigned int s = 0; s < pSteps; s++) {
			double dW = Sqrtdt * hiprand_normal(&pStates[idx]);
			double dS = (pr * S * dt) + (pv * S * dW);
			S += dS;
		}

		WX = S;
		WMOld = WM;
		WM = WM + (WX - WM) / static_cast<double>(i + 1);
		WS = WS + (WX - WM) * (WX - WMOld);

		Sum += S;
		Sum2 += S * S;
		SumError += abs(S - pActual);
	}	

	pSum[idx] = Sum;
	pSum2[idx] = Sum2;
	pVariance[idx] = WS / static_cast<double>(pSims - 1);
	pError[idx] = SumError;
	pSamples[idx] = static_cast<double>(pSims);
}



void Crash(std::string pFile, int pLine, hipError_t pCUDAError, std::string pMessage) {

	std::cerr << "Crash in file " << pFile << std::endl
		<< "Line: " << pLine << std::endl;

	if (pCUDAError != NULL) {
		std::cerr << "CUDA Error: " << pCUDAError << " :: " << hipGetErrorString(pCUDAError) << std::endl;
	}

	exit(_CRASH_);
}
